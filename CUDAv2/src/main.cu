#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include "utils.h"
#include "customTimer.h"
#include "printer.h"
#include "Fraction.h"

Fraction* execHost(StartArgs args)
{
    Timer::getInstance().clear();
    int totalSize=sizeof(Fraction) * args.SIZE(), i;
    Printer bytePrinter("host.data",args);
    void* space,*result= new Fraction[totalSize];
    auto params = initParams();
    if (NULL == result)
    {
        printf("Malloc problem!\n");
        exit(-1);
    }

    space = initSpace(args);

    printf("Host simulation started\n");
    Timer::getInstance().start("Host simulation time");
    for(i=0;i<args.NUM_OF_ITERATIONS;++i)
    {
    	hostSimulation(args,&params,space,result);
		swapPointers(space,result);
		if(args.print)//print iteration if save flag is on
			if(i % 2==0)
				bytePrinter.printIteration((Fraction*)space, i);
			else
				bytePrinter.printIteration((Fraction*)result, i);
    }
    Timer::getInstance().stop("Host simulation time");
    printf("Host simulation completed\n");
    Timer::getInstance().printResults();

    if(false==args.print)//print only last iteration if save flag is off
		if(i % 2==0)
			bytePrinter.printIteration((Fraction*)space, i);
		else
			bytePrinter.printIteration((Fraction*)result, i);

    if(i % 2 == 0)
    {
    	free(result);
		return (Fraction*)space;
    }
    else
    {
    	free(space);
		return (Fraction*)result;
    }
}

Fraction* execDeviceSurface(StartArgs args,FluidParams* params,Fraction* space)
{
    float* floats =spaceToFloats(args,space);
    Printer bytePrinter("device.data",args);

    // For float we could create a channel with:
    hipChannelFormatDesc channelDesc =hipCreateChannelDesc<float>();

    // Allocate memory in device
    hipExtent extent = make_hipExtent(args.X_SIZE*5,args.Y_SIZE,args.Z_SIZE);
    hipArray* cuSpaceArray;
    hipMalloc3DArray(&cuSpaceArray, &channelDesc,extent,hipArraySurfaceLoadStore);
    hipArray* cuResultArray;
    hipMalloc3DArray(&cuResultArray, &channelDesc,extent,hipArraySurfaceLoadStore);

    // Copy to device memory initial data
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr((void*)floats, args.X_SIZE*sizeof(float)*5, args.Y_SIZE, args.Z_SIZE);
    copyParams.dstArray = cuSpaceArray;
    copyParams.extent = extent;
    copyParams.kind	= hipMemcpyHostToDevice;

    hipMemcpy3D(&copyParams);

    // Specify surface
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;

    // Create the surface objects
    resDesc.res.array.array = cuSpaceArray;
    hipSurfaceObject_t spaceSurfObj=0;
    hipCreateSurfaceObject(&spaceSurfObj, &resDesc);
    resDesc.res.array.array = cuResultArray;
    hipSurfaceObject_t resultSurfObj=0;
    hipCreateSurfaceObject(&resultSurfObj, &resDesc);

    void *resultObjPointer=&resultSurfObj,*spaceObjPointer=&spaceSurfObj;
    int i=0;

    printf("Simulation started\n");
    Timer::getInstance().start("Device simulation time");

    for(;i<args.NUM_OF_ITERATIONS;++i)
    {
    	simulationSurface(args,params,*(hipSurfaceObject_t*)spaceObjPointer,*(hipSurfaceObject_t*)resultObjPointer);
		swapPointers(spaceObjPointer,resultObjPointer);

		if(args.print)//print iteration if save flag is on
		{
			copyParams = {0};
			copyParams.extent = extent;
			copyParams.dstPtr = make_hipPitchedPtr((void*)floats, args.X_SIZE*sizeof(float)*5, args.Y_SIZE, args.Z_SIZE);
			copyParams.kind	= hipMemcpyDeviceToHost;
			if(i % 2 == 0)
			{
				copyParams.srcArray = cuSpaceArray;
				hipMemcpy3D(&copyParams);
			}
			else
			{
				copyParams.srcArray = cuResultArray;
				hipMemcpy3D(&copyParams);
			}
			floatsToSpace(args,floats,space);
			bytePrinter.printIteration(space, i);
		}
    }
    if(false==args.print)
    {
		copyParams = {0};
		copyParams.extent = extent;
		copyParams.dstPtr = make_hipPitchedPtr((void*)floats, args.X_SIZE*sizeof(float)*5, args.Y_SIZE, args.Z_SIZE);
		copyParams.kind	= hipMemcpyDeviceToHost;
		if(i % 2 == 0)
		{
			copyParams.srcArray = cuSpaceArray;
			hipMemcpy3D(&copyParams);
		}
		else
		{
			copyParams.srcArray = cuResultArray;
			hipMemcpy3D(&copyParams);
		}
    }
    Timer::getInstance().stop("Device simulation time");
    printf("Simulation completed\n");
    Timer::getInstance().printResults();

    // Destroy surface objects
    hipDestroySurfaceObject(spaceSurfObj);
    hipDestroySurfaceObject(resultSurfObj);

    // Free device memory
    hipFreeArray(cuSpaceArray);
    hipFreeArray(cuResultArray);

    floatsToSpace(args,floats,space);

    if(false==args.print)//print only last iteration if save flag is off
		bytePrinter.printIteration((Fraction*)space, i);

    //free temporary memory
    free(floats);

    return space;
}

Fraction* execDevice(StartArgs args)
{
    Fraction* space = initSpace(args);
    Printer bytePrinter("device.data",args);
    FluidParams *d_params,params = initParams();
    hipMalloc((void **)&d_params, sizeof(FluidParams));
    hipMemcpy(d_params, &params, sizeof(FluidParams), hipMemcpyHostToDevice);

    if(NULL==space)
        exit(-1);

    //DUE TO PROBLEMS WITH POINTERS AND SURFACE MEMORY OBJECTS THIS KIND OF SIMULATION IS THREATED SEPARATELY
    if(SURFACE==args.type)
        return execDeviceSurface(args, d_params,space);
    void *d_space,*d_result;
    int totalSize = sizeof(Fraction)*args.SIZE(), i;
    void *result = new Fraction[totalSize];

    hipMalloc((void **)&d_space,totalSize);
    hipMalloc((void **)&d_result,totalSize);
    cudaCheckErrors("Mallocs");
    hipMemcpy(d_space,space,totalSize, hipMemcpyHostToDevice);
    hipMemcpy(d_result, result, totalSize, hipMemcpyHostToDevice);

    cudaCheckErrors("Copy mem");

    printf("Simulation started\n");
    Timer::getInstance().start("Device simulation time");

    for(i=0;i<args.NUM_OF_ITERATIONS;++i)
    {
    	simulation(args,d_params,d_space,d_result);
		swapPointers(d_space,d_result);

		if(args.print)//print iteration if save flag is on
		{
			if(i % 2 == 0)
				hipMemcpy(space, d_space, totalSize, hipMemcpyDeviceToHost);
			else
				hipMemcpy(space, d_result, totalSize, hipMemcpyDeviceToHost);
			bytePrinter.printIteration(space, i);
		}
		else
			hipDeviceSynchronize();
    }
    if(!args.print)
    {
		if(i % 2 == 0)
			hipMemcpy(space, d_space, totalSize, hipMemcpyDeviceToHost);
		else
			hipMemcpy(space, d_result, totalSize, hipMemcpyDeviceToHost);
		//print only last iteration if save flag is off
		bytePrinter.printIteration((Fraction*)space, i);
    }

    Timer::getInstance().stop("Device simulation time");
    printf("Simulation completed\n");
    Timer::getInstance().printResults();

    hipFree(d_params);
    hipFree(d_space);
    hipFree(d_result);

    return space;
}

void printHelp()
{
	printf("hydro [options]\n"
			"--device <type>\trun simulation on GPU, types:\n"
			"\t\tGLOBAL, SURFACE, SHARED, SHARED_FOR\n"
			"--host\t\trun simulation on CPU\n"
			"--X <size>\tX space size\n"
			"--Y <size>\tY space size\n"
			"--Z <size>\tZ space size\n"
			"--ITER <num>\tnumber of iterations\n"
			"--save\t\tsave all iterations to file, default only last\n"
			"--help\t\tdisplays this massage\n"
			"--random\tstarting space values are random\n"
			"if there is no arguments provided default values are:\n"
			"\thydro --device GLOBAL --X 100 --Y 100 --Z 100 --ITER 10\n");
}

StartArgs parsInputArguments(const int argc, char *argv[])
{
	bool error=false;
	StartArgs args;
	int x,y,z,iter_num;

	//default simulation settings
	args.NUM_OF_ITERATIONS = 10;
	args.X_SIZE = 100;
	args.Y_SIZE = 100;
	args.Z_SIZE = 100;
	args.host = false;
	args.type = GLOBAL;
	args.print = false;
	args.random = false;

	for(int i=1; i<argc; ++i)
	{
		if(strcmp(argv[i],"--device") == 0)
		{
			if((i+1)<argc && strcmp(argv[i+1],"GLOBAL") == 0)
				args.type = GLOBAL;
			else if((i+1)<argc && strcmp(argv[i+1],"SURFACE") == 0)
				args.type = SURFACE;
			else if((i+1)<argc && strcmp(argv[i+1],"SHARED") == 0)
				args.type = SHARED_3D_LAYER;
			else if((i+1)<argc && strcmp(argv[i+1],"SHARED_FOR") == 0)
				args.type = SHARED_3D_LAYER_FOR_IN;
			else
			{
				error = true;
				break;
			}
			++i;
		}
		else if(strcmp(argv[i],"--host") == 0)
		{
			args.host = true;
		}
		else if(strcmp(argv[i],"--random") == 0)
		{
			args.host = true;
		}
		else if(strcmp(argv[i],"--X") == 0)
		{
			if((i+1)<argc && sscanf(argv[i+1],"%d",&x) == 1)
				args.X_SIZE = x;
			else
			{
				error = true;
				break;
			}
			++i;
		}
		else if(strcmp(argv[i],"--Y") == 0)
		{
			if((i+1)<argc && sscanf(argv[i+1],"%d",&y) == 1)
				args.Y_SIZE = y;
			else
			{
				error = true;
				break;
			}
			++i;
		}
		else if(strcmp(argv[i],"--Z") == 0)
		{
			if((i+1)<argc && sscanf(argv[i+1],"%d",&z) == 1)
				args.Z_SIZE = z;
			else
			{
				error = true;
				break;
			}
			++i;
		}
		else if(strcmp(argv[i],"--ITER") == 0)
		{
			if((i+1)<argc && sscanf(argv[i+1],"%d",&iter_num) == 1)
				args.NUM_OF_ITERATIONS = iter_num;
			else
			{
				error = true;
				break;
			}
			++i;
		}
		else if(strcmp(argv[i],"--save") == 0)
		{
			args.print = true;
		}
		else if(strcmp(argv[i],"--help") == 0)
		{
			printHelp();
			exit(EXIT_SUCCESS);
		}
		else
		{
			error=true;
			break;
		}
	}

	if(error)
	{
		printf("ERROR on parsing input arguments, starting simulation with default arguments\n");
		printHelp();
	}

	return args;
}

int main(int argc, char *argv[])
{
    Fraction* hostOutputSpace,* deviceOutputSpace;

    StartArgs args = parsInputArguments(argc,argv);
    initCuda();
    deviceOutputSpace = execDevice(args);

    if(args.host)
    {
        hostOutputSpace = execHost(args);
        compare_results(args,hostOutputSpace,deviceOutputSpace);
        free(hostOutputSpace);
    }

    free(deviceOutputSpace);
    return 0;
}
