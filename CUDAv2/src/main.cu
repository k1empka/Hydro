#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#include "utils.h"
#include "customTimer.h"
#include "printer.h"
#include "Fraction.h"

#define RANDOM false
#define PRINT_RESULTS false

Fraction* execHost()
{
    Timer::getInstance().clear();
    int totalSize=sizeof(Fraction) * SIZE, i;
    void* space,*result= new Fraction[totalSize];
    auto params = initParams();
    if (NULL == result)
    {
        printf("Malloc problem!\n");
        exit(-1);
    }

    space = initSpace(RANDOM);
#if PRINT_RESULTS
    Printer bytePrinter("host.data");
#endif
    printf("Host simulation started\n");
    Timer::getInstance().start("Host simulation time");
    for(i=0;i<NUM_OF_ITERATIONS;++i)
    {
    	hostSimulation(&params,space,result);
		swapPointers(space,result);
#if PRINT_RESULTS
		if(i % 2==0)
			bytePrinter.printIteration((Fraction*)space, i);
		else
			bytePrinter.printIteration((Fraction*)result, i);
#endif
    }
    Timer::getInstance().stop("Host simulation time");
    printf("Host simulation completed\n");
    Timer::getInstance().printResults();

    if(i % 2 == 0)
    {
    	free(result);
		return (Fraction*)space;
    }
    else
    {
    	free(space);
		return (Fraction*)result;
    }
}

Fraction* execDeviceSurface(FluidParams* params,Fraction* space)
{
    float* floats =spaceToFloats(space);

    // For float we could create a channel with:
    hipChannelFormatDesc channelDesc =hipCreateChannelDesc<float>();

    // Allocate memory in device
    hipExtent extent = make_hipExtent(X_SIZE*5,Y_SIZE,Z_SIZE);
    hipArray* cuSpaceArray;
    hipMalloc3DArray(&cuSpaceArray, &channelDesc,extent,hipArraySurfaceLoadStore);
    hipArray* cuResultArray;
    hipMalloc3DArray(&cuResultArray, &channelDesc,extent,hipArraySurfaceLoadStore);

    // Copy to device memory initial data
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr((void*)floats, X_SIZE*sizeof(float)*5, Y_SIZE, Z_SIZE);
    copyParams.dstArray = cuSpaceArray;
    copyParams.extent = extent;
    copyParams.kind	= hipMemcpyHostToDevice;

    hipMemcpy3D(&copyParams);

    // Specify surface
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;

    // Create the surface objects
    resDesc.res.array.array = cuSpaceArray;
    hipSurfaceObject_t spaceSurfObj=0;
    hipCreateSurfaceObject(&spaceSurfObj, &resDesc);
    resDesc.res.array.array = cuResultArray;
    hipSurfaceObject_t resultSurfObj=0;
    hipCreateSurfaceObject(&resultSurfObj, &resDesc);

    void *resultObjPointer=&resultSurfObj,*spaceObjPointer=&spaceSurfObj;
    int i=0;

#if PRINT_RESULTS
    Printer bytePrinter("device.data");
#endif
    printf("Simulation started\n");
    Timer::getInstance().start("Device simulation time");

    for(;i<NUM_OF_ITERATIONS;++i)
    {
    	simulationSurface(params,*(hipSurfaceObject_t*)spaceObjPointer,*(hipSurfaceObject_t*)resultObjPointer);
		swapPointers(spaceObjPointer,resultObjPointer);

#if PRINT_RESULTS
		copyParams = {0};
		copyParams.extent = extent;
		copyParams.dstPtr = make_hipPitchedPtr((void*)floats, X_SIZE*sizeof(float)*5, Y_SIZE, Z_SIZE);
		copyParams.kind	= hipMemcpyDeviceToHost;
        if(i % 2 == 0)
        {
        	copyParams.srcArray = cuSpaceArray;
			hipMemcpy3D(&copyParams);
        }
        else
        {
        	copyParams.srcArray = cuResultArray;
			hipMemcpy3D(&copyParams);
        }
        floatsToSpace(floats,space);
        bytePrinter.printIteration(space, i);
#endif
    }
#if !PRINT_RESULTS
    copyParams = {0};
    copyParams.extent = extent;
	copyParams.dstPtr = make_hipPitchedPtr((void*)floats, X_SIZE*sizeof(float)*5, Y_SIZE, Z_SIZE);
	copyParams.kind	= hipMemcpyDeviceToHost;
    if(i % 2 == 0)
    {
    	copyParams.srcArray = cuSpaceArray;
		hipMemcpy3D(&copyParams);
    }
	else
	{
		copyParams.srcArray = cuResultArray;
		hipMemcpy3D(&copyParams);
	}
#endif
    Timer::getInstance().stop("Device simulation time");
    printf("Simulation completed\n");
    Timer::getInstance().printResults();

    // Destroy surface objects
    hipDestroySurfaceObject(spaceSurfObj);
    hipDestroySurfaceObject(resultSurfObj);

    // Free device memory
    hipFreeArray(cuSpaceArray);
    hipFreeArray(cuResultArray);

    floatsToSpace(floats,space);

    //free temporary memory
    free(floats);

    return space;
}

Fraction* execDevice(enum deviceSimulationType type)
{
    Fraction* space = initSpace(RANDOM);
    FluidParams *d_params,params = initParams();
    hipMalloc((void **)&d_params, sizeof(FluidParams));
    hipMemcpy(d_params, &params, sizeof(FluidParams), hipMemcpyHostToDevice);

    if(NULL==space)
        exit(-1);

    //DUE TO PROBLEMS WITH POINTERS AND SURFACE MEMORY OBJECTS THIS KIND OF SIMULATION IS THREATED SEPARATELY
    if(SURFACE==type)
        return execDeviceSurface(d_params,space);
    void *d_space,*d_result;
    int totalSize = sizeof(Fraction)*SIZE, i;
    void *result = new Fraction[totalSize];

    hipMalloc((void **)&d_space,totalSize);
    hipMalloc((void **)&d_result,totalSize);
    cudaCheckErrors("Mallocs");
    hipMemcpy(d_space,space,totalSize, hipMemcpyHostToDevice);
    hipMemcpy(d_result, result, totalSize, hipMemcpyHostToDevice);

    cudaCheckErrors("Copy mem");
#if PRINT_RESULTS
    Printer bytePrinter("device.data");
#endif
    printf("Simulation started\n");
    Timer::getInstance().start("Device simulation time");

    for(i=0;i<NUM_OF_ITERATIONS;++i)
    {
    	simulation(d_params,d_space,d_result,type);
		swapPointers(d_space,d_result);

#if PRINT_RESULTS
		if(i % 2 == 0)
			hipMemcpy(space, d_space, totalSize, hipMemcpyDeviceToHost);
		else
			hipMemcpy(space, d_result, totalSize, hipMemcpyDeviceToHost);
        bytePrinter.printIteration(space, i);
#else
        hipDeviceSynchronize();
#endif
    }
#if !PRINT_RESULTS
    if(i % 2 == 0)
		hipMemcpy(space, d_space, totalSize, hipMemcpyDeviceToHost);
	else
		hipMemcpy(space, d_result, totalSize, hipMemcpyDeviceToHost);
#endif
    Timer::getInstance().stop("Device simulation time");
    printf("Simulation completed\n");
    Timer::getInstance().printResults();

    hipFree(d_params);
    hipFree(d_space);
    hipFree(d_result);
    return space;
}

int main()
{
    bool hostSimulationOn = true;
	//bool hostSimulationOn = false;
    //enum deviceSimulationType type = GLOBAL;
    //enum deviceSimulationType type = SHARED_3D_LAYER_FOR_IN;
    enum deviceSimulationType type = SURFACE;

    Fraction* hostOutputSpace,* deviceOutputSpace;

    initCuda();
    deviceOutputSpace = execDevice(type);

    if(hostSimulationOn)
    {
        hostOutputSpace = execHost();
        compare_results(hostOutputSpace,deviceOutputSpace);
        free(hostOutputSpace);
    }

    free(deviceOutputSpace);
    return 0;
}
