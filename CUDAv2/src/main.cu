#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include "utils.h"
#include "customTimer.h"
#include "printer.h"
#include "Fraction.h"

#define RANDOM false
#define PRINT_RESULTS false

Fraction* execHost()
{
	Timer::getInstance().clear();
	int totalSize=sizeof(Fraction) * SIZE;
	void* space,*result=(Fraction*)malloc(totalSize);
    if (NULL == result)
    {
        printf("Malloc problem!\n");
        exit(-1);
    }

	space = initSpace(RANDOM);
#if PRINT_RESULTS
    Printer bytePrinter("host.data");
#endif
	printf("Host simulation started\n");
    Timer::getInstance().start("Host simulation time");
	for(int i=0;i<NUM_OF_ITERATIONS;++i)
	{
		if((i % 2) != 0)
		{
			swapPointers(space,result);
		}
		hostSimulation(initParams(),space,result);
#if PRINT_RESULTS
        bytePrinter.printIteration(result, i);
#endif
	}
    Timer::getInstance().stop("Host simulation time");
	printf("Host simulation completed\n");
	Timer::getInstance().printResults();

	free(space);
	return (Fraction*)result;
}

Fraction* execDeviceSurface(Fraction* space)
{
	int memSize=sizeof(float)*SIZE;

	// For float we could create a channel with:
	hipChannelFormatDesc channelDesc =hipCreateChannelDesc(32, 0, 0, 0,hipChannelFormatKindFloat);

	// Allocate memory in device
	hipArray* cuSpaceArray;
	hipMallocArray(&cuSpaceArray, &channelDesc, SIZE,hipArraySurfaceLoadStore);
	hipArray* cuResultArray;
	hipMallocArray(&cuResultArray, &channelDesc, SIZE,hipArraySurfaceLoadStore);

	// Copy to device memory initial data
	hipMemcpyToArray(cuSpaceArray, 0, 0, space, memSize,hipMemcpyHostToDevice);

	// Specify surface
	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;

	// Create the surface objects
	resDesc.res.array.array = cuSpaceArray;
	hipSurfaceObject_t spaceSurfObj=0;
	hipCreateSurfaceObject(&spaceSurfObj, &resDesc);
	resDesc.res.array.array = cuResultArray;
	hipSurfaceObject_t resultSurfObj=0;
	hipCreateSurfaceObject(&resultSurfObj, &resDesc);

#if PRINT_RESULTS
    Printer bytePrinter("device.data");
#endif
	printf("Simulation started\n");
    Timer::getInstance().start("Device simulation time");

	for(int i=0;i<NUM_OF_ITERATIONS;++i)
	{
		if(i%2!=0)
		{
			simulationSurface(resultSurfObj,spaceSurfObj);
#if PRINT_RESULTS
		hipMemcpyFromArray(space,cuSpaceArray, 0, 0, memSize,hipMemcpyDeviceToHost);
        bytePrinter.printIteration(space, i);
#endif
		}
		else
		{
			simulationSurface(spaceSurfObj,resultSurfObj);
#if PRINT_RESULTS
		hipMemcpyFromArray(space,cuResultArray, 0, 0, memSize,hipMemcpyDeviceToHost);
        bytePrinter.printIteration(space, i);
#endif
		}
	}
#if !PRINT_RESULTS
	hipMemcpyFromArray(space,cuResultArray, 0, 0, memSize,hipMemcpyDeviceToHost);
#endif
    Timer::getInstance().stop("Device simulation time");
	printf("Simulation completed\n");
	Timer::getInstance().printResults();

	// Destroy surface objects
	hipDestroySurfaceObject(spaceSurfObj);
	hipDestroySurfaceObject(resultSurfObj);

	// Free device memory
	hipFreeArray(cuSpaceArray);
	hipFreeArray(cuResultArray);

	return space;
}

Fraction* execDevice(enum deviceSimulationType type)
{
	Fraction* space = initSpace(RANDOM);

	if(NULL==space)
		exit(-1);

	//DUE TO PROBLEMS WITH POINTERS AND SURFACE MEMORY OBJECTS THIS KIND OF SIMULATION IS THREATED SEPARATELY
	if(SURFACE==type)
		return execDeviceSurface(space);
    FluidParams params = initParams();
	void *d_space,*d_result,*d_params;
	int totalSize = sizeof(Fraction)*SIZE;
	hipMalloc((void **)&d_space,totalSize);
	hipMalloc((void **)&d_result,totalSize);
    hipMalloc((void **)&d_params, sizeof(FluidParams));
	hipMemcpy(d_space,space,totalSize, hipMemcpyHostToDevice);
    hipMemcpy(d_params, &params, sizeof(FluidParams), hipMemcpyHostToDevice);

#if PRINT_RESULTS
    Printer bytePrinter("device.data");
#endif
	printf("Simulation started\n");
    Timer::getInstance().start("Device simulation time");

	for(int i=0;i<NUM_OF_ITERATIONS;++i)
	{
		if((i % 2) != 0)
		{
			swapPointers(d_space,d_result);
		}
		simulation(d_params,d_space,d_result,type);
#if PRINT_RESULTS
        hipMemcpy(space, d_result, totalSize, hipMemcpyDeviceToHost);
        bytePrinter.printIteration(space, i);
#else
        hipDeviceSynchronize();
#endif
	}
#if !PRINT_RESULTS
    hipMemcpy(space, d_result, totalSize, hipMemcpyDeviceToHost);
#endif
    Timer::getInstance().stop("Device simulation time");
	printf("Simulation completed\n");
	Timer::getInstance().printResults();

	hipFree(d_space);
	hipFree(d_result);
	return space;
}
int main()
{
	bool hostSimulationOn = true;
	enum deviceSimulationType type = GLOBAL;

	Fraction* hostOutputSpace,* deviceOutputSpace;

	initCuda();
	deviceOutputSpace = execDevice(type);

	if(hostSimulationOn)
	{
		hostOutputSpace = execHost();
	}
    compare_results(hostOutputSpace,deviceOutputSpace);
	free(hostOutputSpace);
	free(deviceOutputSpace);
	return 0;
}
