#include <hip/hip_runtime_api.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include "computation.cuh"
#include "Timer.h"

void initCuda()
{
	int nDevices;
    int devCount = hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++)
    {
        hipDeviceProp_t props;
        checkCudaErrors(hipGetDeviceProperties(&props, i));
        printf("CUDA device [%s] has %d Multi-Processors\n",
               props.name, props.multiProcessorCount);
     }
    if(nDevices > 1)
        hipSetDevice(1); //Dla mnie bo mam SLI;
}

fraction* initSpace()
{
	fraction* space = (fraction*)malloc(sizeof(fraction));

	if(NULL==space)
	{
		printf("memory allocation error\n");
		return NULL;
	}

	for(int y=0; y<Y_SIZE; ++y)
		for(int x=0; x<X_SIZE;++x)
		{
			space->Vx[IDX_2D(x,y)]=0.;
			space->Vy[IDX_2D(x,y)]=0.;
			space->U[IDX_2D(x,y)]=0.;
		}

	srand (time(NULL));

	for(int x=0; x<20; ++x)
		for(int y=0; y<20; ++y)

	{
		int idx = IDX_2D(40+x,40+y);
		space->U[idx]= (float)(rand()%MAX_START_FORCE + 1);
		space->Vx[idx]= (float)(rand()%MAX_START_FORCE + 1 - MAX_START_FORCE/2) * 0.05;
		space->Vy[idx]= (float)(rand()%MAX_START_FORCE + 1 - MAX_START_FORCE/2) * 0.01;
	}

	return space;
}

void printHeader(FILE* f)
{
	fprintf(f,"%d %d %d\n",X_SIZE,Y_SIZE,NUM_OF_ITERATIONS);
}

void printIteration(FILE* f,fraction* space, int iter)
{
	fprintf(f,"ITER_%d\n",iter);

	for(int y=0; y<Y_SIZE;++y)
		for(int x=0; x<X_SIZE;++x)
		{
			if(space->U[y*X_SIZE+x] > 0.001f)
				fprintf(f,"%d %d %f %f\n",x,y,space->U[y*X_SIZE+x],space->U[y*X_SIZE+x]);
		}
}

FILE* initOutputFile()
{
	char filename[100];
	sprintf(filename,"result");
	FILE *f = fopen(filename, "w");
	if (f == NULL)
	{
	    printf("Error opening file!\n");
	    exit(1);
	}
	return f;
}

int main()
{
	initCuda();

	fraction* space = initSpace();

	if(NULL==space)
		return -1;

	fraction *d_space,*d_result;
	int totalSize=sizeof(fraction);
	hipMalloc((void **)&d_space,totalSize);
	hipMalloc((void **)&d_result,totalSize);
	hipMemcpy(d_space,space,totalSize, hipMemcpyHostToDevice);

	FILE* f = initOutputFile();

	printHeader(f);
	printf("Simulation started\n");
	for(int i=0;i<NUM_OF_ITERATIONS;++i)
	{
		fraction* tmp;
		Timer::getInstance().start("Simulation time");

		if((i % 2) != 0)
		{
			tmp = d_space;
			d_space = d_result;
			d_result = tmp;
		}

		simulation(d_space,d_result);
		hipMemcpy(space,d_result,totalSize, hipMemcpyDeviceToHost);
		Timer::getInstance().stop("Simulation time");
		printIteration(f,space,i);
	}
	printf("Simulation completed\n");
	Timer::getInstance().printResults();
	hipFree(d_space);
	hipFree(d_result);
	free(space);

	fclose(f);

	return 0;
}
