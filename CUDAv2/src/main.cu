#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#include "utils.h"
#include "customTimer.h"
#include "printer.h"
#include "Fraction.h"

#define RANDOM false
#define PRINT_RESULTS true

Fraction* execHost()
{
    Timer::getInstance().clear();
    int totalSize=sizeof(Fraction) * SIZE;
    void* space,*result= new Fraction[totalSize];
    auto params = initParams();
    if (NULL == result)
    {
        printf("Malloc problem!\n");
        exit(-1);
    }

    space = initSpace(RANDOM);
#if PRINT_RESULTS
    Printer bytePrinter("host.data");
#endif
    printf("Host simulation started\n");
    Timer::getInstance().start("Host simulation time");
    for(int i=0;i<NUM_OF_ITERATIONS;++i)
    {
        if((i % 2) != 0)
        {
            swapPointers(space,result);
        }
        hostSimulation(&params,space,result);
#if PRINT_RESULTS
        bytePrinter.printIteration((Fraction*)result, i);
#endif
    }
    Timer::getInstance().stop("Host simulation time");
    printf("Host simulation completed\n");
    Timer::getInstance().printResults();

    free(space);
    return (Fraction*)result;
}

Fraction* execDeviceSurface(FluidParams* params,Fraction* space)
{
    const int memSize=sizeof(float)*SIZE*5;
    float* floats =spaceToFloats(space);

    // For float we could create a channel with:
    hipChannelFormatDesc channelDesc =hipCreateChannelDesc<float>();

    // Allocate memory in device
    hipArray* cuSpaceArray;
    hipMallocArray(&cuSpaceArray, &channelDesc, SIZE*5,hipArraySurfaceLoadStore);
    hipArray* cuResultArray;
    hipMallocArray(&cuResultArray, &channelDesc, SIZE*5,hipArraySurfaceLoadStore);

    // Copy to device memory initial data
    hipMemcpyToArray(cuSpaceArray, 0, 0, floats, memSize,hipMemcpyHostToDevice);

    // Specify surface
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;

    // Create the surface objects
    resDesc.res.array.array = cuSpaceArray;
    hipSurfaceObject_t spaceSurfObj=0;
    hipCreateSurfaceObject(&spaceSurfObj, &resDesc);
    resDesc.res.array.array = cuResultArray;
    hipSurfaceObject_t resultSurfObj=0;
    hipCreateSurfaceObject(&resultSurfObj, &resDesc);

    void *resultObjPointer=&resultSurfObj,*spacetObjPointer=&spaceSurfObj;
    int i=0;

#if PRINT_RESULTS
    Printer bytePrinter("device.data");
#endif
    printf("Simulation started\n");
    Timer::getInstance().start("Device simulation time");

    for(;i<NUM_OF_ITERATIONS;++i)
    {
        if((i % 2) != 0)
        {
            swapPointers(spacetObjPointer,resultObjPointer);
        }
        simulationSurface(params,*(hipSurfaceObject_t*)spacetObjPointer,*(hipSurfaceObject_t*)resultObjPointer);
#if PRINT_RESULTS
        if(i % 2 != 0) 
            hipMemcpyFromArray(floats,cuSpaceArray, 0, 0, memSize,hipMemcpyDeviceToHost);
        else 
            hipMemcpyFromArray(floats,cuResultArray, 0, 0, memSize,hipMemcpyDeviceToHost);
#endif
    }
#if !PRINT_RESULTS
    if(i%2!=0) 
        hipMemcpyFromArray(floats,cuSpaceArray, 0, 0, memSize,hipMemcpyDeviceToHost);
    else 
        hipMemcpyFromArray(floats,cuResultArray, 0, 0, memSize,hipMemcpyDeviceToHost);
#endif
    Timer::getInstance().stop("Device simulation time");
    printf("Simulation completed\n");
    Timer::getInstance().printResults();

    // Destroy surface objects
    hipDestroySurfaceObject(spaceSurfObj);
    hipDestroySurfaceObject(resultSurfObj);

    // Free device memory
    hipFreeArray(cuSpaceArray);
    hipFreeArray(cuResultArray);

    floatsToSpace(floats,space);

    //free temporary memory
    free(floats);

    return space;
}

Fraction* execDevice(enum deviceSimulationType type)
{
    Fraction* space = initSpace(RANDOM);
    FluidParams *d_params,params = initParams();
    hipMalloc((void **)&d_params, sizeof(FluidParams));
    hipMemcpy(d_params, &params, sizeof(FluidParams), hipMemcpyHostToDevice);

    if(NULL==space)
        exit(-1);

    //DUE TO PROBLEMS WITH POINTERS AND SURFACE MEMORY OBJECTS THIS KIND OF SIMULATION IS THREATED SEPARATELY
    if(SURFACE==type)
        return execDeviceSurface(d_params,space);
    void *d_space,*d_result;
    int totalSize = sizeof(Fraction)*SIZE;
    hipMalloc((void **)&d_space,totalSize);
    hipMalloc((void **)&d_result,totalSize);
    cudaCheckErrors("Mallocs");
    hipMemcpy(d_space,space,totalSize, hipMemcpyHostToDevice);
    cudaCheckErrors("Copy mem");
#if PRINT_RESULTS
    Printer bytePrinter("device.data");
#endif
    printf("Simulation started\n");
    Timer::getInstance().start("Device simulation time");

    for(int i=0;i<NUM_OF_ITERATIONS;++i)
    {
        if((i % 2) != 0)
        {
            swapPointers(d_space,d_result);
        }
        simulation(d_params,d_space,d_result,type);
#if PRINT_RESULTS
        hipMemcpy(space, d_result, totalSize, hipMemcpyDeviceToHost);
        bytePrinter.printIteration(space, i);
#else
        hipDeviceSynchronize();
#endif
    }
#if !PRINT_RESULTS
    hipMemcpy(space, d_result, totalSize, hipMemcpyDeviceToHost);
#endif
    Timer::getInstance().stop("Device simulation time");
    printf("Simulation completed\n");
    Timer::getInstance().printResults();

    hipFree(d_params);
    hipFree(d_space);
    hipFree(d_result);
    return space;
}
int main()
{
    bool hostSimulationOn = true;
    enum deviceSimulationType type = GLOBAL;

    Fraction* hostOutputSpace,* deviceOutputSpace;

    initCuda();
    deviceOutputSpace = execDevice(type);

    if(hostSimulationOn)
    {
        hostOutputSpace = execHost();
        compare_results(hostOutputSpace,deviceOutputSpace);
        free(hostOutputSpace);
    }
    free(deviceOutputSpace);
    return 0;
}
