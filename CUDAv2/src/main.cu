#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include "utils.h"
#include "customTimer.h"
#include "printer.h"

#define RANDOM false
#define PRINT_RESULTS false

fraction* execHost()
{
	Timer::getInstance().clear();
	int totalSize=sizeof(fraction);
	void* space,*result=(fraction*)malloc(totalSize);
    if (NULL == result)
    {
        printf("Malloc problem!\n");
        exit(-1);
    }

	space = initSpace(RANDOM);
#if PRINT_RESULTS
    Printer bytePrinter("host.data");
#endif
	printf("Host simulation started\n");
    Timer::getInstance().start("Host simulation time");
	for(int i=0;i<NUM_OF_ITERATIONS;++i)
	{
		if((i % 2) != 0)
		{
			swapPointers(space,result);
		}
		hostSimulation(space,result);
#if PRINT_RESULTS
        bytePrinter.printIteration(result, i);
#endif
	}
    Timer::getInstance().stop("Host simulation time");
	printf("Host simulation completed\n");
	Timer::getInstance().printResults();

	free(space);
	return (fraction*)result;
}

fraction* execDevice(enum deviceSimulationType type)
{
	fraction* space = initSpace(RANDOM);

	if(NULL==space)
		exit(-1);

	void *d_space,*d_result;
	int totalSize = sizeof(fraction);
	hipMalloc((void **)&d_space,totalSize);
	hipMalloc((void **)&d_result,totalSize);
	hipMemcpy(d_space,space,totalSize, hipMemcpyHostToDevice);

#if PRINT_RESULTS
    Printer bytePrinter("device.data");
#endif
	printf("Simulation started\n");
    Timer::getInstance().start("Device simulation time");

	for(int i=0;i<NUM_OF_ITERATIONS;++i)
	{
		if((i % 2) != 0)
		{
			swapPointers(d_space,d_result);
		}
		simulation(d_space,d_result,type);
#if PRINT_RESULTS
        hipMemcpy(space, d_result, totalSize, hipMemcpyDeviceToHost);
        bytePrinter.printIteration(space, i);
#endif
	}
#if !PRINT_RESULTS
    hipMemcpy(space, d_result, totalSize, hipMemcpyDeviceToHost);
#endif
    Timer::getInstance().stop("Device simulation time");
	printf("Simulation completed\n");
	Timer::getInstance().printResults();

	hipFree(d_space);
	hipFree(d_result);
	return space;
}
int main()
{
	bool hostSimulationOn = true;
	enum deviceSimulationType type = GLOBAL;

	fraction* hostOutputSpace,* deviceOutputSpace;

	initCuda();
	deviceOutputSpace = execDevice(type);

	if(hostSimulationOn)
	{
		hostOutputSpace = execHost();
	}
    compare_results(hostOutputSpace,deviceOutputSpace);
	free(hostOutputSpace);
	free(deviceOutputSpace);
	return 0;
}
