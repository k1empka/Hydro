#include "hip/hip_runtime.h"
#include "computation.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while(0);


__device__ float bilinterp(float* source, float x, float y,int xSize, int ySize)
{
	int x0,x1,y0,y1;
	float fx0,fx1,fy0,fy1;

	//Boundaries
	if(x > float(xSize)-2.0+0.5)
		x = float(xSize)-2.0+0.5;
	if(y > float(ySize)-2.0+0.5)
		y = float(ySize)-2.0+0.5;
	if(x < 0.5f)
		x = 0.5f;
	if(y < 0.5f)
		y = 0.5f;

	x0 = int(x);
	x1 = x0 + 1;
	y0 = int(y);
	y1 = y0 + 1;

	fx1 = (float)x - x0;
	fx0 = (float)1 - fx1;
	fy1 = (float)y - y0;
	fy0 = (float)1 - fy1;

	return 		   (float) fx0 * (fy0 * source[IDX_2D(x0,y0)]  +
						          fy1 * source[IDX_2D(x0,y1)]) +
					       fx1 * (fy0 * source[IDX_2D(x1,y0)]  +
					    		  fy1 * source[IDX_2D(x1,y1)]);
}

__global__ void advect(fraction* spaceData,fraction* resultData, float dt)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if(x > 1 && y > 1 && x<X_SIZE && y<Y_SIZE)
	{
		int id = IDX_2D(x,y);
		int newX = float(x) - dt * spaceData->Vx[id];
		int newY = float(y) - dt * spaceData->Vy[id];

		resultData->Vx[id] = bilinterp(spaceData->Vx,newX,newY,X_SIZE,Y_SIZE);
		resultData->Vy[id] = bilinterp(spaceData->Vy,newX,newY,X_SIZE,Y_SIZE);
		resultData->U[id]  = bilinterp(spaceData->U, newX,newY,X_SIZE,Y_SIZE);
	}
}


/*				Shared memory model
 * 				 ________________
 * 				|t1				|
 * 			____|t2_____________|_____
 * 		  |t1|t2|t1|t2|			|     |
 * 		  |	    |t2				|	  |
 * 		  |  	|				|	  |
 * 		  | 	|				|	  |
 * 		  |  	|				|	  |
 * 		  |  	|				|	  |
 * 		  |_____|_______________|_____|
 * 				|				|
 * 				|_______________|
 *
 * 			Border threads like t1,t2 copy also memory as described above
 */


__global__ void stepSh(fraction* spaceData,fraction* resultData)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int nCount = 2; //neighbours count
	extern __shared__ float shSpace[];
	shSpace[THX_2D(threadIdx.x,threadIdx.y)] = 0;
	shSpace[THX_2D(threadIdx.x+4,threadIdx.y+4)] = 0;

	if(x<X_SIZE && y<Y_SIZE)
	{
		float* result = resultData->U;
		float* space  = spaceData->U;
		int thx = threadIdx.x+2, thy = threadIdx.y+2;
		int idx = IDX_2D(x,y);

		__syncthreads(); // wait for threads to fill whole shared memory

		shSpace[THX_2D(thx,thy)] = space[IDX_2D(x,y)];

		if(threadIdx.x == 0 && x > 1)
		{
			shSpace[THX_2D(thx - 2,thy)] = space[IDX_2D(x-2,y)];
		}
		if(threadIdx.x == 1 && x > 2)
		{
			shSpace[THX_2D(thx - 2,thy)] = space[IDX_2D(x-2,y)];
		}
		if(threadIdx.x == blockDim.x - nCount && x < X_SIZE - 2)
		{
			shSpace[THX_2D(thx + nCount,thy)] = space[IDX_2D(x+nCount,y)];
		}
		if(threadIdx.x == blockDim.x - 1 && x < X_SIZE - 1)
		{
			shSpace[THX_2D(thx + nCount,thy)] = space[IDX_2D(x+nCount,y)];
		}
		if(threadIdx.y == 0 && y > 1)
		{
			shSpace[THX_2D(thx,thy - nCount)] = space[IDX_2D(x,y-nCount)];
		}
		if(threadIdx.y  == 1 && y > 2)
		{
			shSpace[THX_2D(thx,thy - nCount)] = space[IDX_2D(x,y-nCount)];
		}
		if(threadIdx.y  == blockDim.y - nCount && y < Y_SIZE - 2)
		{
			shSpace[THX_2D(thx,thy + nCount)] = space[IDX_2D(x,y+nCount)];
		}
		if(threadIdx.y  == blockDim.y - 1 && y < Y_SIZE - 1)
		{
			shSpace[THX_2D(thx,thy + nCount)] = space[IDX_2D(x,y+nCount)];
		}

		__syncthreads(); // wait for threads to fill whole shared memory

		result[idx]  = 0.7  * shSpace[THX_2D(thx,thy)];
		result[idx] += 0.05 * shSpace[THX_2D(thx,thy-1)];
		result[idx] += 0.025* shSpace[THX_2D(thx,thy-2)];
		result[idx] += 0.05 * shSpace[THX_2D(thx,thy+1)];
		result[idx] += 0.025* shSpace[THX_2D(thx,thy+2)];
		result[idx] += 0.05 * shSpace[THX_2D(thx-1,thy)];
		result[idx] += 0.025* shSpace[THX_2D(thx-2,thy)];
		result[idx] += 0.05 * shSpace[THX_2D(thx+1,thy)];
		result[idx] += 0.025* shSpace[THX_2D(thx+2,thy)];
	}
}

__global__ void step(fraction* spaceData,fraction* resultData)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if(x<X_SIZE && y<Y_SIZE)
	{
		float* result = resultData->U;
		float* space  = spaceData->U;
		int idx = IDX_2D(x,y);

		result[idx] = 0.7*space[idx];

		if( (y-1) > 0 )
			result[idx] +=.05 *space[(y-1)*X_SIZE+x];
		if( (y-2) > 0 )
			result[idx] +=.025*space[(y-2)*X_SIZE+x];
		if( (y+1) < Y_SIZE )
			result[idx] +=.05 *space[(y+1)*X_SIZE+x];
		if( (y+2) < Y_SIZE )
			result[idx] +=.025*space[(y+2)*X_SIZE+x];
		if( (x-1) > 0 )
			result[idx] +=.05 *space[(y)*X_SIZE+x-1];
		if( (x-2) > 0 )
			result[idx] +=.025*space[(y)*X_SIZE+x-2];
		if( (x+1) < X_SIZE )
			result[idx] +=.05 *space[(y)*X_SIZE+x+1];
		if( (x+2) < X_SIZE )
			result[idx] +=.025*space[(y)*X_SIZE+x+2];
	}
}


void simulation(fraction* d_space,fraction* d_result)
{
	static dim3 threadsPerBlock(TH_IN_BLCK_X, TH_IN_BLCK_Y);
	static dim3 numBlocks(ceil(float(X_SIZE) / float(threadsPerBlock.x)),
						  ceil(float(Y_SIZE) / float(threadsPerBlock.y)));
	static int	shMemSize = sizeof(float) *
		(threadsPerBlock.x + 4) *
		(threadsPerBlock.y + 4); // each thread - each cell);
				// + boundaries threads need neighbours from other block

	//advect<<<numBlocks,threadsPerBlock>>>(d_space,d_result,DT);
	//step<<<numBlocks, threadsPerBlock>>>(d_space,d_result);
	stepSh<<<numBlocks, threadsPerBlock,shMemSize>>>(d_space,d_result);
    cudaCheckErrors("stepSh failed!");
}
