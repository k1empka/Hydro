#include "exec.h"

Fraction* execDeviceSurface(StartArgs args, FluidParams* params, Fraction* space)
{
    float* floats = spaceToFloats(args, space);
    Printer* bytePrinter = NULL;

    // For float we could create a channel with:
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

    // Allocate memory in device
    hipExtent extent = make_hipExtent(args.X_SIZE * 5, args.Y_SIZE, args.Z_SIZE);
    hipArray* cuSpaceArray;
    hipMalloc3DArray(&cuSpaceArray, &channelDesc, extent, hipArraySurfaceLoadStore);
    hipArray* cuResultArray;
    hipMalloc3DArray(&cuResultArray, &channelDesc, extent, hipArraySurfaceLoadStore);

    // Copy to device memory initial data
    hipMemcpy3DParms copyParams = { 0 };
    copyParams.srcPtr = make_hipPitchedPtr((void*)floats, args.X_SIZE * sizeof(float) * 5, args.Y_SIZE, args.Z_SIZE);
    copyParams.dstArray = cuSpaceArray;
    copyParams.extent = extent;
    copyParams.kind = hipMemcpyHostToDevice;

    hipMemcpy3D(&copyParams);

    // Specify surface
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;

    // Create the surface objects
    resDesc.res.array.array = cuSpaceArray;
    hipSurfaceObject_t spaceSurfObj = 0;
    hipCreateSurfaceObject(&spaceSurfObj, &resDesc);
    resDesc.res.array.array = cuResultArray;
    hipSurfaceObject_t resultSurfObj = 0;
    hipCreateSurfaceObject(&resultSurfObj, &resDesc);

    void *resultObjPointer = &resultSurfObj, *spaceObjPointer = &spaceSurfObj;
    int i = 0;

    if (args.print)
        bytePrinter = new Printer("device.data", args);

    printf("Simulation started\n");
    Timer::getInstance().start("Device simulation time");

    for (; i<args.NUM_OF_ITERATIONS; ++i)
    {
        simulationSurface(args, params, *(hipSurfaceObject_t*)spaceObjPointer, *(hipSurfaceObject_t*)resultObjPointer);
        swapPointers(spaceObjPointer, resultObjPointer);

        if (args.print)
        {
            copyParams = { 0 };
            copyParams.extent = extent;
            copyParams.dstPtr = make_hipPitchedPtr((void*)floats, args.X_SIZE * sizeof(float) * 5, args.Y_SIZE, args.Z_SIZE);
            copyParams.kind = hipMemcpyDeviceToHost;
            if (i % 2 == 0)
            {
                copyParams.srcArray = cuSpaceArray;
                hipMemcpy3D(&copyParams);
            }
            else
            {
                copyParams.srcArray = cuResultArray;
                hipMemcpy3D(&copyParams);
            }
            floatsToSpace(args, floats, space);
            bytePrinter->printIteration(space, i);
        }
    }
    if (!args.print)
    {
        copyParams = { 0 };
        copyParams.extent = extent;
        copyParams.dstPtr = make_hipPitchedPtr((void*)floats, args.X_SIZE * sizeof(float) * 5, args.Y_SIZE, args.Z_SIZE);
        copyParams.kind = hipMemcpyDeviceToHost;
        if (i % 2 == 0)
        {
            copyParams.srcArray = cuSpaceArray;
            hipMemcpy3D(&copyParams);
        }
        else
        {
            copyParams.srcArray = cuResultArray;
            hipMemcpy3D(&copyParams);
        }
    }
    Timer::getInstance().stop("Device simulation time");
    printf("Simulation completed\n");
    Timer::getInstance().printResults();

    // Destroy surface objects
    hipDestroySurfaceObject(spaceSurfObj);
    hipDestroySurfaceObject(resultSurfObj);

    // Free device memory
    hipFreeArray(cuSpaceArray);
    hipFreeArray(cuResultArray);

    floatsToSpace(args, floats, space);

    //free temporary memory
    free(floats);

    if (bytePrinter) delete bytePrinter;

    return space;
}

Fraction* execDevice(StartArgs args)
{
    Fraction* space = initSpace(args);
    Printer* bytePrinter = NULL;
    FluidParams *d_params, params = initParams();
    hipMalloc((void **)&d_params, sizeof(FluidParams));
    hipMemcpy(d_params, &params, sizeof(FluidParams), hipMemcpyHostToDevice);

    if (NULL == space)
        exit(-1);

    //DUE TO PROBLEMS WITH POINTERS AND SURFACE MEMORY OBJECTS THIS KIND OF SIMULATION IS THREATED SEPARATELY
    if (deviceSimulationType::SURFACE == args.type)
        return execDeviceSurface(args, d_params, space);
    void *d_space, *d_result;
    int totalSize = sizeof(Fraction)*args.SIZE(), i;
    void *result = new Fraction[totalSize];

    hipMalloc((void **)&d_space, totalSize);
    hipMalloc((void **)&d_result, totalSize);
    cudaCheckErrors("Mallocs");
    hipMemcpy(d_space, space, totalSize, hipMemcpyHostToDevice);
    hipMemcpy(d_result, result, totalSize, hipMemcpyHostToDevice);

    cudaCheckErrors("Copy mem");
    if (args.print)
        bytePrinter = new Printer("device.data", args);
    printf("Simulation started\n");
    Timer::getInstance().start("Device simulation time");

    for (i = 0; i<args.NUM_OF_ITERATIONS; ++i)
    {
        simulation(args, d_params, d_space, d_result);
        swapPointers(d_space, d_result);

        if (args.print)
        {
            if (i % 2 == 0)
                hipMemcpy(space, d_space, totalSize, hipMemcpyDeviceToHost);
            else
                hipMemcpy(space, d_result, totalSize, hipMemcpyDeviceToHost);
            bytePrinter->printIteration(space, i);
        }
        else
            hipDeviceSynchronize();
    }
    if (!args.print)
        if (i % 2 == 0)
            hipMemcpy(space, d_space, totalSize, hipMemcpyDeviceToHost);
        else
            hipMemcpy(space, d_result, totalSize, hipMemcpyDeviceToHost);

    Timer::getInstance().stop("Device simulation time");
    printf("Simulation completed\n");
    Timer::getInstance().printResults();

    hipFree(d_params);
    hipFree(d_space);
    hipFree(d_result);

    if (bytePrinter) delete bytePrinter;

    return space;
}
